#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>

#define MAX_LENGTH 8
#define CHARSET_SIZE 62
#define THREADS_PER_BLOCK 256
#define TARGET_HASH 0

// Character set: 0-9, a-z, A-Z
__constant__ char charset[CHARSET_SIZE] = {
    '0','1','2','3','4','5','6','7','8','9',
    'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z',
    'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'
};

// Hash function matching the JavaScript implementation
__device__ uint32_t hash_string(const char* str, int len) {
    uint32_t hash = 3735928559U; // Initial value from JS code
    
    for (int i = 0; i < len; i++) {
        hash = ((hash << 5) - hash) + (uint32_t)str[i];
    }
    
    return hash;
}

// Convert thread index to string combination
__device__ void index_to_string(uint64_t index, char* str, int len) {
    for (int i = 0; i < len; i++) {
        str[len - 1 - i] = charset[index % CHARSET_SIZE];
        index /= CHARSET_SIZE;
    }
    str[len] = '\0';
}

// Calculate total combinations for given length
__host__ uint64_t calculate_combinations(int len) {
    uint64_t total = 1;
    for (int i = 0; i < len; i++) {
        total *= CHARSET_SIZE;
    }
    return total;
}

__global__ void brute_force_kernel(int string_length, uint64_t start_index, uint64_t total_combinations, int* found, char* result) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t global_idx = start_index + idx;
    
    if (global_idx >= total_combinations) return;
    
    char test_string[MAX_LENGTH + 1];
    index_to_string(global_idx, test_string, string_length);
    
    uint32_t hash = hash_string(test_string, string_length);
    
    if (hash == TARGET_HASH) {
        if (atomicCAS(found, 0, 1) == 0) {
            // First thread to find the solution
            for (int i = 0; i <= string_length; i++) {
                result[i] = test_string[i];
            }
        }
    }
}

void brute_force_length(int len) {
    printf("Searching strings of length %d...\n", len);
    
    uint64_t total_combinations = calculate_combinations(len);
    printf("Total combinations: %llu\n", total_combinations);
    
    int* d_found;
    char* d_result;
    int h_found = 0;
    char h_result[MAX_LENGTH + 1] = {0};
    
    hipMalloc(&d_found, sizeof(int));
    hipMalloc(&d_result, MAX_LENGTH + 1);
    hipMemcpy(d_found, &h_found, sizeof(int), hipMemcpyHostToDevice);
    
    uint64_t batch_size = 1000000000ULL; // Process in batches to avoid timeout
    
    for (uint64_t start = 0; start < total_combinations; start += batch_size) {
        uint64_t end = min(start + batch_size, total_combinations);
        uint64_t current_batch_size = end - start;
        
        int blocks = (current_batch_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        
        brute_force_kernel<<<blocks, THREADS_PER_BLOCK>>>(len, start, total_combinations, d_found, d_result);
        hipDeviceSynchronize();
        
        // Check if found
        hipMemcpy(&h_found, d_found, sizeof(int), hipMemcpyDeviceToHost);
        if (h_found) {
            hipMemcpy(h_result, d_result, MAX_LENGTH + 1, hipMemcpyDeviceToHost);
            printf("\n=== SUCCESS! ===\n");
            printf("Found input that hashes to 0: '%s'\n", h_result);
            
            // Verify on host
            uint32_t verify_hash = 3735928559U;
            for (int i = 0; i < len; i++) {
                verify_hash = ((verify_hash << 5) - verify_hash) + (uint32_t)h_result[i];
            }
            printf("Verification hash: %u\n", verify_hash);
            printf("=== SUCCESS! ===\n\n");
            
            hipFree(d_found);
            hipFree(d_result);
            
            printf("Press Enter to exit...");
            getchar();
            return;
        }
        
        printf("Processed %llu/%llu combinations...\n", end, total_combinations);
    }
    
    printf("No solution found for length %d\n", len);
    hipFree(d_found);
    hipFree(d_result);
}

int main() {
    printf("CUDA Hash Brute Force - Target: %u\n", TARGET_HASH);
    printf("Using character set: %s\n", "0-9, a-z, A-Z");
    
    // Check CUDA device
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using device: %s\n", prop.name);
    
    // Test the hash function with known input
    printf("\nTesting with '324e2':\n");
    uint32_t test_hash = 3735928559U;
    const char* test_str = "324e2";
    for (int i = 0; i < 5; i++) {
        test_hash = ((test_hash << 5) - test_hash) + (uint32_t)test_str[i];
    }
    printf("Hash of '324e2': %u\n", test_hash);
    
    // Start brute force search
    printf("\nStarting brute force search...\n");
    
    for (int len = 1; len <= MAX_LENGTH; len++) {
        brute_force_length(len);
    }
    
    printf("\nSearch completed. No solution found within length %d.\n", MAX_LENGTH);
    printf("Press Enter to exit...");
    getchar();
    
    return 0;
}

// Compilation command:
// nvcc -o hash_bruteforce hash_bruteforce.cu -arch=sm_60